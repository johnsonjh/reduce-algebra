#include "hip/hip_runtime.h"
// mr.cu                                      Copyright (C) A C Norman 2017

/*
 * "This software contains source code provided by NVIDIA Corporation."
 *
 * Specifically this code contains fragments from the CUDA Sample code
 * distributed by nVidia. They permit modification and re-distribution subject
 * to including the statement made at the top of this comments and preserving
 * nVidia Copyright statements, such as the following line:
 * "Copyright 1993-2015 NVIDIA Corporation.  All rights reserved."
 *
 * Apart from the above  constraints the code here is subject to a BSD
 * license, the terms of which follow:
 */


/**************************************************************************
 * Copyright (C) 2017, Codemist.                         A C Norman       *
 *                                                                        *
 * Redistribution and use in source and binary forms, with or without     *
 * modification, are permitted provided that the following conditions are *
 * met:                                                                   *
 *                                                                        *
 *     * Redistributions of source code must retain the relevant          *
 *       copyright notice, this list of conditions and the following      *
 *       disclaimer.                                                      *
 *     * Redistributions in binary form must reproduce the above          *
 *       copyright notice, this list of conditions and the following      *
 *       disclaimer in the documentation and/or other materials provided  *
 *       with the distribution.                                           *
 *                                                                        *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS    *
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT      *
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS      *
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE         *
 * COPYRIGHT OWNERS OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,   *
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,   *
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS  *
 * OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND *
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR  *
 * TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF     *
 * THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH   *
 * DAMAGE.                                                                *
 *************************************************************************/

#define __STDC_CONSTANT_MACROS 1
#define __STDC_LIMIT_MACROS    1
#define __STDC_FORMAT_MACROS   1

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <inttypes.h>
#include <stdint.h>
#include <time.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

__global__ void gpuCode(uint32_t n, char *out)
{
// n will be a composite.
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
// Ignore calls that seem out of range.
    if (bx>=16 || by>=16 || tx>=16 || ty>=16) return;
    uint32_t a = ((bx*16 + by)*16 + tx)*16 + ty;
    uint32_t a0 = a;
// a is a base I might like to use to test it. I will set out[a] to 1
// if the Miller Rabin test suggests that n is prime, ie if n is a strong
// pseudoprime base a.
// When this is called n should not be a multiple of 2, 3, 5, 7 or 11, and so
// the next few (commented out) lines are not required.
//  if (n == 1) goto composite;            // 1 treated as composite
//  else if (n <= 3) goto prime;           // 2 and 3 are primes
//  else if ((n & 1) == 0) goto composite; // even numbers are all composite
// a == 1 and a == p-1 (and for small p values that are close to multiples of
// p) will always be bad, but it is easiest to let those cases through here.
    uint32_t d = n-1;
    int s = 0;
    while ((d & 1) == 0)
    {   d = d >> 1;
        s++;
    }
    uint32_t y = 1;
    while (d > 1)
    {   if ((d & 1) != 0) y = ((uint64_t)a * y) % n;
        a = ((uint64_t)a * a) % n;
        d = d >> 1;
    }
    uint32_t x = ((uint64_t)a * y) % n;
    if (x == 1 || x == n-1) goto prime;
    while (s > 1)
    {   x = ((uint64_t)x * x) % n;
        if (x == 1) goto composite;
        else if (x == n-1) goto prime;
        s--;
    }
// If I drop out of the loop then the input was compostite.
composite:
    out[a0] = 0;
    return;
prime:
    out[a0] = 1;
    return;
}

// I will report progress from time to time. More often at the start of the
// run.

static char *get_date()
{   time_t t0 = time(NULL);
    return asctime(localtime(&t0));
}

unsigned int records = 0;

void report_progress(uint32_t p)
{   if ((p < 0x00100000 && (p & 0x0000fffe) == 0) ||
        (p < 0x01000000 && (p & 0x000ffffe) == 0) ||
        (p < 0x10000000 && (p & 0x00fffffe) == 0) ||
        (                  (p & 0x0ffffffe) == 0))
    {   printf("Testing %.8x found %u at %s", p&(~1), records, get_date());
        fflush(stdout);
    }
}

static void dump_results(uint32_t p, char *out_data, FILE *out)
{   uint32_t fails = 0;
// Bases 0 and 1 are not useful at all. p-1 is also no use
    out_data[0] = out_data[1] = 0;
    for (int i=0; i<256*256; i++)
        if (out_data[i]) fails++;
    if (fails != 0)
    {   records++;
// For each composite number that is a pseudo-prime to at least one base I
// output a record that starts with 4 bytes for the composite uint32_t value.
        putc(p & 0xff, out);
        putc((p>>8) & 0xff, out);
        putc((p>>16) & 0xff, out);
        putc((p>>24) & 0xff, out);
// The 2 bytes for the number of bases (in the range 1..0xffff) that it is
// a pseudoprime for.
        putc(fails & 0xff, out);
        putc((fails>>8) & 0xff, out);
// Then a collection of 2 byte values that list those bases.
        for (int i=0; i<256*256; i+=8)
        {   if (out_data[i])
            {   putc(i & 0xff, out);
                putc((i>>8) & 0xff, out);
            }
        }
    }
}

int main(int argc, char *argv[])
{   FILE *out = fopen("mr.data", "wb");
    if (out == NULL)
    {   printf("Failed to open output file\n");
        exit(1);
    }
    setvbuf(out, NULL, _IOFBF, 4*1024*1024);
//
// I will use 256 Mbytes for a bitmap recording all the odd primes up to 2^32.
// It takes (only) a couple of minutes of CPU time to set this up.
//
    char *primes = (char *)malloc(0x80000000U/8);
    if (primes == NULL)
    {   printf("Unable to allocate space\n");
        exit(1);
    }
// Sieve method
    printf("Start prime sieve %s", get_date());
    fflush(stdout);
    for (uint64_t i=0; i<0x80000000U/8; i++) primes[i] = 0xff;
    primes[0] = 0xfe;
    {   uint64_t p = 1;
        while (p*p < UINT64_C(0x100000000))
        {   while ((primes[p/16] & (1<<((p/2)&7))) == 0) p+=2; // next prime.
            uint64_t np = (3*p-1)/2;
            while (np < 0x80000000U)   // strike out all multiple of it.
            {   primes[np/8] &= ~(1<<(np&7));
                np += p;
            }
            p += 2;
        }
    }
    printf("Map of odd primes created %s", get_date());
// Both for checking and foir use elsewhere I will dump out a brief bitmap
// table of primes that will let me identify primes up to 4096 very rapidly.
    printf("uint32_t oddprime_bitmap[] =");
    for (int i=0; i<4096/2/8/4; i++)
    {   if (i == 0) printf("\n{   ");
        else if (i % 6 == 0) printf(",\n    ");
        else printf(", ");
        uint32_t v = primes[4*i] | (primes[4*i+1]<<8) |
                     (primes[4*i+2]<<16) | (primes[4*i+3]<<24);
        printf("%#.8x", v);
    }
    printf("\n};\n");
    fflush(stdout);

    int devID = gpuGetMaxGflopsDeviceId();
    checkCudaErrors(hipSetDevice(devID));

// While debugging I wanted to use printf on the device, and giving a
// large print buffer helped. For release code this is not needed.
//  hipDeviceSetLimit(hipLimitPrintfFifoSize, 200*1024*1024);

    char *out_data = (char *)malloc(256*256*sizeof(char));
    int out_ready = 0;
    uint32_t saved_p = 0;

    char *gpu_output;
    checkCudaErrors(hipMalloc((void **)&gpu_output, 256*256*sizeof(char)));

    dim3 grid(16, 16);
    dim3 thread(16, 16);

    records = 0;
    for (uint32_t p=3; p!=0xffffffff; p+=2)
    {   report_progress(p);
        if (p%3 == 0) continue;
        if (p%5 == 0) continue;
        if (p%7 == 0) continue;
        if (p%11 == 0) continue;
// I am not going to test numbers that have a factor of 2,3,5,7 or 11 because
// I will check for those small factors elsewhere. I then just need to look
// at other composite numbers to see if they might be strong pseudoprimes.
        if ((primes[p/16] & (1<<((p/2)&7))) != 0) continue;
// Now p should be a composite number (with no factors less than 13).
//
// Start the GPU...
        gpuCode<<<grid, thread>>>(p, gpu_output);
// Now if a previous kernel task has given us some output write it to
// our output file. What I am doing here is overlapping the GPU analysing
// my next number with the CPU processing and writing out information about
// the one I analysed last time round.
        if (out_ready) dump_results(saved_p, out_data, out);
// When that is done I should wait for the current kernel to complete
// and recover the results that it delivers.
        getLastCudaError("Kernel execution failed");
        checkCudaErrors(hipMemcpy(out_data,
                                   gpu_output,
                                   256*256*sizeof(char),
                                   hipMemcpyDeviceToHost));
        saved_p = p;
        out_ready = 1;
    }
// The final chunk of output from the GPU has now been recovered but still
// needs sending to the output file.
    if (out_ready) dump_results(saved_p, out_data, out);

    fclose(out);
    free(out_data);
    checkCudaErrors(hipFree(gpu_output));

    exit(EXIT_SUCCESS);
}

// end of mr.cu
